#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/ATen.h>
#include <hip/hip_runtime.h>
#include <vector>

// example of index_max_kernel
__global__ void index_max_cuda_kernel(
    const float* index_vals,
    const int* indices,
    int A_num_block,
    int B_num_block,
    float* output) {

    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // For simplicity, an example where it performs element-wise max
    if (idx < A_num_block * B_num_block) {
        output[idx] = max(index_vals[idx], (float)indices[idx]);
    }
}

// host function that wraps the kernel
at::Tensor index_max_kernel(
    at::Tensor index_vals,
    at::Tensor indices,
    int A_num_block,
    int B_num_block) {

    // allocate output tensor
    auto output = at::zeros({A_num_block, B_num_block}, index_vals.options());

 
    int threads = 1024;
    int blocks = (A_num_block * B_num_block + threads - 1) / threads;
    index_max_cuda_kernel<<<blocks, threads>>>(
        index_vals.data_ptr<float>(),
        indices.data_ptr<int>(),
        A_num_block,
        B_num_block,
        output.data_ptr<float>());

    return output;
}
